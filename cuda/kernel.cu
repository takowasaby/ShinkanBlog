#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <float.h>

#include "hip/hip_runtime.h"

#define CHECK_CUDA_ERROR(call)																									\
{																																\
	const hipError_t error = call;																								\
	if (error != hipSuccess)																									\
	{																															\
		char str[1024];																											\
		sprintf_s(str, 1024, "Cuda Error: %s:%d, cuda:%d, reason: %s\n", __FILE__, __LINE__, error, hipGetErrorString(error));	\
		return str;																												\
	}																															\
}																																\

namespace cuda
{
#if 0
}   // indent guard
#endif

constexpr size_t BLOCK_SIZE = 32;

__device__ __host__ void updateObjectPosition(Object & object, double updateTime)
{
	object.centerX_ += object.velocityX_ * updateTime;
	object.centerY_ += object.velocityY_ * updateTime;
}

__device__ __host__ void updateObjectVelocity(Object & object, double updateTime)
{
	object.velocityX_ += object.accelerationX_ * updateTime;
	object.velocityY_ += object.accelerationY_ * updateTime;
}

__device__ __host__ bool checkCollisionObjects(const Object & obj1, const Object & obj2, double deltatime, double & collisionTime)
{
	double Dx0 = obj1.centerX_ - obj2.centerX_;
	double Dy0 = obj1.centerY_ - obj2.centerY_;

	double Dx1 = obj1.centerX_ - obj2.centerX_ + obj1.velocityX_ - obj2.velocityX_;
	double Dy1 = obj1.centerY_ - obj2.centerY_ + obj1.velocityY_ - obj2.velocityY_;

	double dDx = Dx1 - Dx0;
	double dDy = Dy1 - Dy0;

	double P = dDx * dDx + dDy * dDy;
	double Q = dDx * Dx0 + dDy * Dy0;
	double R = Dx0 * Dx0 + Dy0 * Dy0;

	double r12S = (obj1.radius_ + obj2.radius_) * (obj1.radius_ + obj2.radius_);

	if (R < r12S)
	{
		collisionTime = 0.0;
		return true;
	}

	if (P < DBL_EPSILON)
	{
		return false;
	}

	double judge = Q * Q - P * (R - r12S);

	if (judge < 0)
	{
		return false;
	}

	double aqrtJudge = sqrt(judge);
	double T1 = (-Q + judge) / P;
	double T2 = (-Q - judge) / P;
	double T = T1 < T2 ? T1 : T2;

	if (T < 0 || deltatime < T)
	{
		return false;
	}

	collisionTime = T;

	return true;
}

__device__ __host__ bool checkCollisionObjectLine(const Object & obj, const LineObject & line, double deltatime, double & collisionTime)
{
	double Nx = line.y1_ - line.y2_;
	double Ny = line.x1_ - line.x2_;

	double NN = sqrt(Nx * Nx + Ny * Ny);

	if (NN < DBL_EPSILON)
	{
		return false;
	}

	double PN = abs((obj.centerX_ - line.x1_) * Nx + (obj.centerY_ - line.y1_) * Ny);
	double VN = obj.velocityX_ * Nx + obj.velocityY_ * Ny;

	double D = PN / NN - obj.radius_;
	double dD = VN / NN;

	if (dD * dD < DBL_EPSILON)
	{
		return false;
	}

	double T = D / dD;

	if (T < 0 || deltatime < T)
	{
		return false;
	}

	collisionTime = T;

	return true;
}

__device__ __host__ void collisionProcessObjects(Object & obj1, Object & obj2, const double objectMass, const double coefficientOfRestitution)
{
	double Dx = obj2.centerX_ - obj1.centerX_;
	double Dy = obj2.centerY_ - obj1.centerY_;

	if (Dx < DBL_EPSILON && Dx < DBL_EPSILON)
	{
		return;
	}

	double nDx = Dx / sqrt(Dx * Dx + Dy * Dy);
	double nDy = Dy / sqrt(Dx * Dx + Dy * Dy);

	double VD12x = obj1.velocityX_ - obj2.velocityX_;
	double VD12y = obj1.velocityY_ - obj2.velocityY_;
	double VD21x = obj2.velocityX_ - obj1.velocityX_;
	double VD21y = obj2.velocityY_ - obj1.velocityY_;

	double S1 = VD21x * nDx + VD21y * nDy;
	double S2 = VD12x * nDx + VD12y * nDy;

	if (objectMass < DBL_EPSILON)
	{
		return;
	}

	double C1 = (objectMass / (objectMass + objectMass)) * (1 + coefficientOfRestitution) * S1;
	double C2 = (objectMass / (objectMass + objectMass)) * (1 + coefficientOfRestitution) * S2;

	obj1.velocityX_ = C1 * nDx + obj1.velocityX_;
	obj1.velocityY_ = C1 * nDy + obj1.velocityY_;
	obj2.velocityX_ = C2 * nDx + obj2.velocityX_;
	obj2.velocityY_ = C2 * nDy + obj2.velocityY_;
}

__device__ __host__ void collisionProcessObjectLine(Object & obj, const LineObject & line, const double objectMass, const double coefficientOfRestitution)
{
	double Nx = line.y1_ - line.y2_;
	double Ny = line.x1_ - line.x2_;

	if (Nx * Nx < DBL_EPSILON && Ny * Ny < DBL_EPSILON)
	{
		return;
	}

	double nNx = Nx / sqrt(Nx * Nx + Ny * Ny);
	double nNy = Ny / sqrt(Nx * Nx + Ny * Ny);

	double dotNV = obj.velocityX_ * nNx + obj.velocityY_ * nNy;

	double C = (1 + coefficientOfRestitution) * dotNV;

	obj.velocityX_ = obj.velocityX_ - C * nNx;
	obj.velocityY_ = obj.velocityY_ - C * nNy;
}

__global__ void updateObjects(double deltatime, Object* objSrc, Object* objDst, size_t objSIze, LineObject* lnObjSrc, size_t lnObjSize, const double objectMass, const double coefficientOfRestitution)
{
	const size_t objIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	Object obj = objSrc[objIdx];
	updateObjectVelocity(obj, deltatime);
	for (size_t i = 0; i < 4; i++)
	{
		double collisionTime = 0.0;
		if (checkCollisionObjectLine(obj, lnObjSrc[i], deltatime, collisionTime))
		{
			updateObjectPosition(obj, collisionTime);
			collisionProcessObjectLine(obj, lnObjSrc[i], objectMass, coefficientOfRestitution);
			updateObjectPosition(obj, deltatime - collisionTime);
			objDst[objIdx] = obj;
			return;
		}
	}
	const size_t halfObjSize = (objSIze + 1) / 2;
	const size_t endCheckIdx = (objIdx + halfObjSize + 1) % objSIze;
	Object otherObj;
	for (size_t i = objIdx + 1; i != endCheckIdx; i = (i + 1) % objSIze)
	{
		double collisionTime = 0.0;
		otherObj = objSrc[i];
		if (cuda::checkCollisionObjects(obj, otherObj, deltatime, collisionTime))
		{
			cuda::updateObjectPosition(obj, collisionTime);
			cuda::updateObjectPosition(otherObj, collisionTime);
			cuda::collisionProcessObjects(obj, otherObj, objectMass, coefficientOfRestitution);
			cuda::updateObjectPosition(obj, deltatime - collisionTime);
			// cuda::updateObjectPosition(otherObj, deltatime - collisionTime);
			objDst[objIdx] = obj;
			return;
		}
	}
	updateObjectPosition(obj, deltatime);
	objDst[objIdx] = obj;
	return;
}

char* gpu_update(double deltatime, Object* objects, size_t objectsSize, LineObject* lineObjects, size_t lineObjectsSize, const double objectMass, const double coefficientOfRestitution)
{
	Object* dInObjects;
	LineObject* dInLineObjects;

	Object* dOutObjects;

	CHECK_CUDA_ERROR(hipHostMalloc((void**)&dInObjects, sizeof(Object) * objectsSize));
	CHECK_CUDA_ERROR(hipHostMalloc((void**)&dInLineObjects, sizeof(LineObject) * lineObjectsSize));

	CHECK_CUDA_ERROR(hipHostMalloc((void**)&dOutObjects, sizeof(Object) * objectsSize));

	CHECK_CUDA_ERROR(hipMemcpy(dInObjects, objects, sizeof(Object) * objectsSize, hipMemcpyHostToDevice));
	CHECK_CUDA_ERROR(hipMemcpy(dInLineObjects, lineObjects, sizeof(LineObject) * lineObjectsSize, hipMemcpyHostToDevice));

	dim3 block(BLOCK_SIZE);
	dim3 grid((objectsSize + block.x - 1) / block.x);

	updateObjects <<<grid, block>>>(deltatime, dInObjects, dOutObjects, objectsSize, dInLineObjects, lineObjectsSize, objectMass, coefficientOfRestitution);

	CHECK_CUDA_ERROR(hipDeviceSynchronize());
	
	CHECK_CUDA_ERROR(hipMemcpy(objects, dOutObjects, sizeof(Object) * objectsSize, hipMemcpyDeviceToHost));

	CHECK_CUDA_ERROR(hipHostFree((void*)dInObjects));
	CHECK_CUDA_ERROR(hipHostFree((void*)dInLineObjects));
	CHECK_CUDA_ERROR(hipHostFree((void*)dOutObjects));

	return nullptr;
}

}
